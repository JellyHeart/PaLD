#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define TILE_WIDTH 32


__global__ void matmul(float* A, float* B, float* C, int width){
	//using shared memory method
	__shared__ float submatrix_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float submatrix_B[TILE_WIDTH][TILE_WIDTH];
	
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	int row = bx*TILE_WIDTH + tx;
	int col = by*TILE_WIDTH + ty;
	float temp=0;
	
	
	for(int phase=0; phase < ceil(1.0*width/TILE_WIDTH); phase++){
		
		//Prepare shared memory at each phase
		if ( row < width && phase*TILE_WIDTH+ty < width)
			submatrix_A[tx][ty] = A[row*width + phase*TILE_WIDTH+ty];
		else 
			submatrix_A[tx][ty] = 0;

		if (col < width &&  phase*TILE_WIDTH +tx < width)
			submatrix_B[tx][ty] = B[(phase*TILE_WIDTH +tx)*width + col]; 
		else
			submatrix_B[tx][ty] = 0;
	
		__syncthreads();

		// calculation within a phase
		if (row < width && col < width)
			for(int i = 0; i<TILE_WIDTH; i++)
				temp += submatrix_A[tx][i] * submatrix_B[i][ty];

		__syncthreads();
	}
	//write back values
	if (row<width && col <width)
		C[row*width + col] = temp;
}

int main(int argc, char** argv){
	int width, i;
	
	if (argc!=2 || !(width=atoi(argv[1])) || width<=0){
		fprintf(stderr,"Please enter a valid width!\n");
		exit(-1);
	}

	//allocating space
	u_int32_t num_elements = width*width;
	u_int32_t matrix_size = sizeof(float)*num_elements;
	float* temp=(float*)malloc(matrix_size);
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, matrix_size);
	hipMalloc(&d_B, matrix_size);
	hipMalloc(&d_C, matrix_size);

	srand(0);
	//create and transfer random numbers
	for(i=0;i<num_elements;i++)
		temp[i] = 20.*rand()/RAND_MAX-10;
	hipMemcpy(d_A, temp, matrix_size, hipMemcpyHostToDevice);
	
	for(i=0;i<num_elements;i++)
		temp[i] = 20.*rand()/RAND_MAX-10;
	hipMemcpy(d_B, temp, matrix_size, hipMemcpyHostToDevice);
	
	int block_len = ceil(1.*width/TILE_WIDTH);
	dim3 dimGrid(block_len, block_len);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

	//warm_up
	matmul<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, width);
	hipDeviceSynchronize();

	//initializing timing tools
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//timing
	hipEventRecord(start);
	matmul<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, width);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	//Transfer back and clean up
	hipMemcpy(temp, d_C, matrix_size, hipMemcpyDeviceToHost);
	float msec;
	hipEventElapsedTime(&msec, start, stop);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	
	// outputing
	
	FILE* fp = fopen("product.dat","w+");
	if(fp == NULL){
		fprintf(stderr, "Failed to load output file");
		exit(-1);
	}
	int j;
	for(i=0;i<width;i++){
		for(j=0;j<width-1;j++)
			fprintf(fp,"%-5.2f\t",temp[i * width + j]);
		fprintf(fp,"%-5.2f\n",temp[i * width + width -1]);
	}
	//close files and free electrons
	fclose(fp);
	free(temp);
	printf("Computation finished in %.5f seconds with N = %d \n", msec/1000, width);
	
}
